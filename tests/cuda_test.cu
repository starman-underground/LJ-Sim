#include <iostream>
#include <hip/hip_runtime.h>


__global__ void hello_cuda_kernel() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from CUDA! Index %d, Thread %d, Block %d\n", idx, threadIdx.x, blockIdx.x);
}

__global__ void vector_add_kernel(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    std::cout << "=== CUDA Test Program ===" << std::endl;

    int device_count;
    hipError_t err = hipGetDeviceCount(&device_count);

    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    std::cout << "Number of CUDA devices: " << device_count << std::endl;

    if (device_count == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return 1;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Device: " << prop.name << std::endl;
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Total Global Memory: " << prop.totalGlobalMem << " bytes" << std::endl;
    std::cout << "Shared Memory per Block: " << prop.sharedMemPerBlock << " bytes" << std::endl;
    std::cout << "Registers per Block: " << prop.regsPerBlock << std::endl;
    std::cout << "Warp Size: " << prop.warpSize << std::endl;
    std::cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Max Threads per Block Dim: " << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << std::endl;
    std::cout << "Max Grid Size: " << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << std::endl;

    // Test 1: kernel launch
    std::cout << "\n=== Test 1: Kernel Launch ===" << std::endl;
    hello_cuda_kernel<<<2, 4>>>();
    // Test 2: vector addition
    std::cout << "\n=== Test 2: Vector Addition ===" << std::endl;
    const int n = 1000;
    size_t size = n * sizeof(float);

    // Allocate memory on host
    float* h_a = (float*)malloc(size);
    float* h_b = (float*)malloc(size);
    float* h_c = (float*)malloc(size);

    // Initialize arrays
    for (int i = 0; i < n; i++) {
        h_a[i] = static_cast<float>(i * 1.0f);
        h_b[i] = static_cast<float>(i * 2.0f);
    }

    // Allocate memory on device
    float* d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threads_per_block = 256;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
    vector_add_kernel<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Verify results
    bool success = true;
    for (int i = 0; i < n; i++) {
        float expected = h_a[i] + h_b[i];
        if (std::abs(h_c[i] - expected) > 1e-6) {
            success = false;
            break;
        }
    }

    if (success) {
        std::cout << "Vector addition test passed!" << std::endl;
    } else {
        std::cout << "Vector addition test failed!" << std::endl;
    }

    // Clean up
    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}